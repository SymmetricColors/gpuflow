#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include <stdio.h>
#include <rte_mbuf.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <iostream>
#include <rte_ip.h>
#include "cuda_async_lcore_function.h"
#include "cuda_lpm_factory.h"

namespace gpuflow {
namespace cu {

__device__ void EtherCopy(struct ether_hdr *ether_header, uint8_t port_id, uint8_t dst_port,
                          ether_addr *dev_mac_addresses_array) {
  // Change source address of this port's
  memcpy(&ether_header->s_addr, &dev_mac_addresses_array[port_id], sizeof(ether_addr));
  // Change dst address of dst port's
  memcpy(&ether_header->d_addr, &dev_mac_addresses_array[dst_port], sizeof(ether_addr));
}

__device__ void IPv4Processing(CustomEtherIPHeader *custom_ether_ip_header, IPv4RuleEntry *lpm_table_ptr,
                               uint8_t port_id, ether_addr *dev_mac_addresses_array, uint8_t *dst_port) {

  // Force cast to ipv4 header
  struct ipv4_hdr *ipv4_header = (struct ipv4_hdr *)&custom_ether_ip_header->ipv6_header;

  // Transferring endian, 32-bit
  uint32_t ipv4_addr_le = ((ipv4_header->dst_addr >> 24) & 0xff) |
          ((ipv4_header->dst_addr << 8) & 0xff0000) |
          ((ipv4_header->dst_addr >> 8) & 0xff00) |
          ((ipv4_header->dst_addr << 24) & 0xff000000);

  if ((lpm_table_ptr + (ipv4_addr_le >> 16)) != nullptr) {
    IPv4RuleEntry *entry = (lpm_table_ptr + (ipv4_addr_le >> 16));
    if (entry->valid_flag) {
      printf("Get the next hop! %d\n", entry->next_hop);
      EtherCopy(&custom_ether_ip_header->ether_header, port_id, entry->next_hop, dev_mac_addresses_array);
      *dst_port = entry->next_hop;
    } else {
      *dst_port = 254;
    }
  }
}

__device__ void IPv6Processing(CustomEtherIPHeader *custom_ether_ip_header) {
  printf("Dealing with ipv6 header!\n");
}

__global__ void PacketProcessing(CustomEtherIPHeader *dev_custom_ether_ip_header_ring,
                                 uint8_t port_id,
                                 uint8_t *dev_dst_port_ring,
                                 IPv4RuleEntry *lpm_table_ptr,
                                 ether_addr *dev_mac_addresses_array,
                                 int nb_of_ip_hdrs) {
  int idx = threadIdx.x;
  if (idx < nb_of_ip_hdrs) {
    // Match up packet types.
    if(dev_custom_ether_ip_header_ring[idx].ether_header.ether_type ==
            (((ETHER_TYPE_IPv4 >> 8) | (ETHER_TYPE_IPv4 << 8)) & 0xffff)) {
      // IPv4 header
      IPv4Processing(&dev_custom_ether_ip_header_ring[idx], lpm_table_ptr, port_id, dev_mac_addresses_array,
                     &dev_dst_port_ring[idx]);
    } else if (dev_custom_ether_ip_header_ring[idx].ether_header.ether_type ==
            (((ETHER_TYPE_IPv6 >> 8) | (ETHER_TYPE_IPv6 << 8)) & 0xffff)) {
      // IPv6 header
      IPv6Processing(&dev_custom_ether_ip_header_ring[idx]);
      dev_dst_port_ring[idx] = 254;
    } else if (dev_custom_ether_ip_header_ring[idx].ether_header.ether_type ==
            (((ETHER_TYPE_ARP >> 8) | (ETHER_TYPE_ARP << 8)) & 0xffff)){
      printf("Arp!\n");
      // Send to all
      dev_dst_port_ring[idx] = 255;
    }
  }
}

static inline void CudaMallocWithFailOver(void **predicate, size_t size, const char *predicate_type) {
  hipError_t error = hipMalloc(predicate, size);
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation on " << predicate_type << " failed, abort." << std::endl;
    std::cerr << hipGetErrorName(error) << " " << hipGetErrorString(error) << std::endl;
    exit(1);
  }
}

static inline void CudaASyncMemcpyWithFailOver(void *dst, const void *src, size_t size, hipMemcpyKind kind,
                                       hipStream_t stream, const char *operation_type) {
  hipError_t error = hipMemcpyAsync(dst, src, size, kind, stream);
  if (error != hipSuccess) {
    std::cerr << "Async Memory copy error on " << operation_type << std::endl;
    std::cerr << hipGetErrorName(error) << " " << hipGetErrorString(error) << std::endl;
    exit(1);
  }
}

int CudaASyncLCoreFunction::SetupCudaDevices() {
  CudaMallocWithFailOver((void **) &dev_mac_addresses_array, num_of_eth_devs * sizeof(struct ether_addr),
                         "dev_mac_addresses_array");
  // Copy mac addresses into device memory
  hipStream_t mac_stream;
  hipStreamCreate(&mac_stream);
  unsigned int count = 0;
  for (auto it = mac_addresses_ptr->begin(); it != mac_addresses_ptr->end(); ++it) {
    CudaASyncMemcpyWithFailOver(&dev_mac_addresses_array[count++],
                                &(*it),
                                sizeof(ether_addr),
                                hipMemcpyHostToDevice,
                                mac_stream, "dev_mac_addresses_array_memory_copy");
  }

  CudaMallocWithFailOver((void **) &dev_custom_ether_ip_headers_ring, 256 * sizeof(CustomEtherIPHeader),
                         "dev_custom_ether_ip_headers_ring");
  CudaMallocWithFailOver((void **) &dev_dst_ports_ring, 256 * sizeof(uint8_t), "dev_dst_ports_ring");
  hipDeviceSynchronize();
  return 0;
}

int CudaASyncLCoreFunction::ProcessPacketsBatch(struct rte_mbuf **pkts_burst, int nb_rx,
                                                IPv4RuleEntry *lpm_table_ptr) {
  hipStream_t hip_stream;
  hipStreamCreate(&hip_stream);
  for (uint8_t i = 0; i < nb_rx; ++i) {
    CudaASyncMemcpyWithFailOver(&dev_custom_ether_ip_headers_ring[head + i],
                                rte_pktmbuf_mtod(pkts_burst[i], struct ether_hdr *),
                                sizeof(CustomEtherIPHeader),
                                hipMemcpyHostToDevice,
                                hip_stream,
                                "custom_ether_ip_header_memory_copy");
  }

  PacketProcessing<<<1, nb_rx, 0, hip_stream>>>(&dev_custom_ether_ip_headers_ring[head],
          port_id,
          &dev_dst_ports_ring[head],
          lpm_table_ptr,
          dev_mac_addresses_array,
          nb_rx);

  for (uint8_t index = 0; index < nb_rx; index++) {
    CudaASyncMemcpyWithFailOver(&host_dst_ports_ring[head + index],
                                &dev_dst_ports_ring[head + index],
                                sizeof(uint8_t),
                                hipMemcpyDeviceToHost,
                                hip_stream,
                                "dev_dst_ports_ring_memory_copy_back");

    CudaASyncMemcpyWithFailOver(rte_pktmbuf_mtod(pkts_burst[index], struct ether_hdr *),
                                &dev_custom_ether_ip_headers_ring[head + index],
                                sizeof(ether_hdr),
                                hipMemcpyDeviceToHost,
                                hip_stream,
                                "custom_ether_header_memory_copy_back");
  }

  // Move on
  head += (uint8_t) nb_rx;
  tail += (uint8_t) nb_rx;
  if (tail == head) {
    std::cerr << "Weird error occured, tail == head" << std::endl;
    exit(1);
  }
  // Sync
  hipStreamSynchronize(hip_stream);
  hipDeviceSynchronize();

  uint8_t local_head = head - (uint8_t) nb_rx;
  for (uint8_t i = 0; i < (uint8_t) nb_rx; i++) {
    struct rte_mbuf *mbuf = pkts_burst[i];
    if (host_dst_ports_ring[local_head + i] == (uint8_t) 255) {
      // Broadcast
      for (uint8_t port = 0; port < num_of_eth_devs; port++) {
        if (port == port_id) {
          continue;
        }
        int send = rte_eth_tx_burst(port, 0, &mbuf, 1);
        if (send > 0) {
          // success
        } else {
          // The drop can't be memory aligned in cuda object.
          // We need to drop at cpp file.
          // Although, it's not that necessary to drop it.
        }
      }
    } else {
      if (host_dst_ports_ring[local_head + i] > (uint8_t) num_of_eth_devs) {
        // Drop out, non configured port.
        continue;
      }
      int send = rte_eth_tx_burst(host_dst_ports_ring[local_head + i], 0, &mbuf, 1);
      if (send > 0) {
        // success
      } else {
        // drop
      }
    }
  }
  return 0;
}

CudaASyncLCoreFunction::CudaASyncLCoreFunction(uint8_t _port_id, unsigned int _num_of_eth_devs,
                                               std::vector<ether_addr> *_mac_addresses_ptr)
        : port_id(_port_id), num_of_eth_devs(_num_of_eth_devs), mac_addresses_ptr(_mac_addresses_ptr),
          head(0), tail(255) {
  // Do nothing
}

} // namespace cu
} // namespace gpuflow

