#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include <stdio.h>
#include <rte_mbuf.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <iostream>
#include <rte_ip.h>
#include "cuda_async_lcore_function.h"
#include "cuda_lpm_factory.h"

namespace gpuflow {
namespace cu {

__device__ void EtherCopy(struct ether_hdr *ether_header, uint8_t port_id, uint8_t dst_port,
                          ether_addr *dev_mac_addresses_array) {
  // Change source address of this port's
  memcpy(&ether_header->s_addr, &dev_mac_addresses_array[port_id], sizeof(ether_addr));
  // Change dst address of dst port's
  memcpy(&ether_header->d_addr, &dev_mac_addresses_array[dst_port], sizeof(ether_addr));
}

__device__ void IPv4Processing(CustomEtherIPHeader *custom_ether_ip_header, IPv4RuleEntry *lpm_table_ptr,
                               uint8_t port_id, ether_addr *dev_mac_addresses_array, uint8_t *dst_port) {

  // Force cast to ipv4 header
  struct ipv4_hdr *ipv4_header = (struct ipv4_hdr *)&custom_ether_ip_header->ipv6_header;

  // Transferring endian, 32-bit
  uint32_t ipv4_addr_le = ((ipv4_header->dst_addr >> 24) & 0xff) |
          ((ipv4_header->dst_addr << 8) & 0xff0000) |
          ((ipv4_header->dst_addr >> 8) & 0xff00) |
          ((ipv4_header->dst_addr << 24) & 0xff000000);

  if ((lpm_table_ptr + (ipv4_addr_le >> 16)) != nullptr) {
    IPv4RuleEntry *entry = (lpm_table_ptr + (ipv4_addr_le >> 16));
    if (entry->valid_flag) {
      EtherCopy(&custom_ether_ip_header->ether_header, port_id, entry->next_hop, dev_mac_addresses_array);
      *dst_port = entry->next_hop;
#ifdef _DEBUG
      printf("Received IPv4 packet, next hop -> %d\n", entry->next_hop);
#endif
    } else {
      *dst_port = 254;
    }
  }
}

__device__ void IPv6Processing(CustomEtherIPHeader *custom_ether_ip_header) {
  // TODO : Add ipv6 processing
}

__global__ void PacketProcessing(CustomEtherIPHeader *dev_custom_ether_ip_header_burst,
                                 uint8_t port_id,
                                 uint8_t *dev_dst_port_burst,
                                 IPv4RuleEntry *lpm_table_ptr,
                                 ether_addr *dev_mac_addresses_array,
                                 int nb_of_ip_hdrs) {
  int idx = threadIdx.x;
  if (idx < nb_of_ip_hdrs) {
    // Match up packet types.
    if(dev_custom_ether_ip_header_burst[idx].ether_header.ether_type ==
            (((ETHER_TYPE_IPv4 >> 8) | (ETHER_TYPE_IPv4 << 8)) & 0xffff)) {
      // IPv4 header
      IPv4Processing(&dev_custom_ether_ip_header_burst[idx], lpm_table_ptr, port_id, dev_mac_addresses_array,
                     &dev_dst_port_burst[idx]);
    } else if (dev_custom_ether_ip_header_burst[idx].ether_header.ether_type ==
            (((ETHER_TYPE_IPv6 >> 8) | (ETHER_TYPE_IPv6 << 8)) & 0xffff)) {
      // IPv6 header
      IPv6Processing(&dev_custom_ether_ip_header_burst[idx]);
      dev_dst_port_burst[idx] = 254;
    } else if (dev_custom_ether_ip_header_burst[idx].ether_header.ether_type ==
            (((ETHER_TYPE_ARP >> 8) | (ETHER_TYPE_ARP << 8)) & 0xffff)){
      // Send to all
      dev_dst_port_burst[idx] = 255;
    }
  }
}

static inline void CudaMallocWithFailOver(void **predicate, size_t size, const char *predicate_type) {
  hipError_t error = hipMalloc(predicate, size);
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation on " << predicate_type << " failed, abort." << std::endl;
    std::cerr << hipGetErrorName(error) << " " << hipGetErrorString(error) << std::endl;
    exit(1);
  }
}

static inline void CudaASyncMemcpyWithFailOver(void *dst, const void *src, size_t size, hipMemcpyKind kind,
                                       hipStream_t stream, const char *operation_type) {
  hipError_t error = hipMemcpyAsync(dst, src, size, kind, stream);
  if (error != hipSuccess) {
    std::cerr << "Async Memory copy error on " << operation_type << std::endl;
    std::cerr << hipGetErrorName(error) << " " << hipGetErrorString(error) << std::endl;
    exit(1);
  }
}

int CudaASyncLCoreFunction::SetupCudaDevices() {
  CudaMallocWithFailOver((void **) &dev_mac_addresses_array, num_of_eth_devs * sizeof(struct ether_addr),
                         "dev_mac_addresses_array");
  // Copy mac addresses into device memory
  hipStream_t mac_stream;
  hipStreamCreate(&mac_stream);
  unsigned int count = 0;
  for (auto it = mac_addresses_ptr->begin(); it != mac_addresses_ptr->end(); ++it) {
    CudaASyncMemcpyWithFailOver(&dev_mac_addresses_array[count++],
                                &(*it),
                                sizeof(ether_addr),
                                hipMemcpyHostToDevice,
                                mac_stream, "dev_mac_addresses_array_memory_copy");
  }


  hipDeviceSynchronize();
  return 0;
}

ProcessingBatchFrame::ProcessingBatchFrame(uint8_t _batch_size) : pkts_burst(nullptr), batch_size(_batch_size),
                                                                  busy(false) {
  for (int i = 0; i < 32; i++) {
    hipStreamCreate(&hip_stream[i]);
  }
  CudaMallocWithFailOver((void **) &dev_custom_ether_ip_headers_burst, batch_size * sizeof(CustomEtherIPHeader),
                         "dev_custom_ether_ip_headers_burst");
  CudaMallocWithFailOver((void **) &dev_dst_ports_burst, batch_size * sizeof(uint8_t), "dev_dst_ports_burst");
}

void CudaASyncLCoreFunction::CreateProcessingBatchFrame(int num_of_batch, uint8_t batch_size){
  batch_head = new ProcessingBatchFrame *[num_of_batch];
  for (int i = 0; i < num_of_batch; i++) {
    batch_head[i] = new ProcessingBatchFrame(batch_size);
  }
}

int CudaASyncLCoreFunction::ProcessPacketsBatch(int batch_idx, struct rte_mbuf **pkts_burst, int nb_rx) {
  auto self_batch = batch_head[batch_idx];
  self_batch->pkts_burst = pkts_burst;
  if (self_batch->busy) {
    std::cout << "Retrieve a busy batch, error occurred, abort." << std::endl;
    return -1;
  }
  self_batch->busy = true;
  for (uint8_t i = 0; i < nb_rx; ++i) {
    CudaASyncMemcpyWithFailOver(&self_batch->dev_custom_ether_ip_headers_burst[i],
                                rte_pktmbuf_mtod(pkts_burst[i], struct ether_hdr *),
                                sizeof(CustomEtherIPHeader),
                                hipMemcpyHostToDevice,
                                self_batch->hip_stream[i],
                                "custom_ether_ip_header_memory_copy");
  }

  for (uint8_t i = 0; i < nb_rx; ++i) {
    PacketProcessing <<< 1, 1, 0, self_batch->hip_stream[i]>>>(&self_batch->dev_custom_ether_ip_headers_burst[i],
            port_id,
            &self_batch->dev_dst_ports_burst[i],
            lpm_table_ptr,
            dev_mac_addresses_array,
            nb_rx);
  }

  for (uint8_t i = 0; i < nb_rx; ++i) {
    CudaASyncMemcpyWithFailOver(&self_batch->host_dst_ports_burst[i], &self_batch->dev_dst_ports_burst[i],
                                sizeof(uint8_t),
                                hipMemcpyDeviceToHost, self_batch->hip_stream[i], "dev_dst_ports_burst_memory_copy_back");
  }

  for (uint8_t index = 0; index < nb_rx; index++) {
    CudaASyncMemcpyWithFailOver(rte_pktmbuf_mtod(self_batch->pkts_burst[index], struct ether_hdr *),
                                &self_batch->dev_custom_ether_ip_headers_burst[index],
                                sizeof(ether_hdr),
                                hipMemcpyDeviceToHost,
                                self_batch->hip_stream[index],
                                "custom_ether_header_memory_copy_back");
  }

  // FIXME: Currently, sync here.
  for (int i = 0; i < nb_rx; ++i) {
    hipStreamSynchronize(self_batch->hip_stream[i]);
  }
  hipDeviceSynchronize();

  for (uint8_t i = 0; i < (uint8_t) nb_rx; i++) {
    struct rte_mbuf *mbuf = self_batch->pkts_burst[i];
    if (self_batch->host_dst_ports_burst[i] == (uint8_t) 255) {
      // Broadcast
      for (uint8_t port = 0; port < num_of_eth_devs; port++) {
        if (port == port_id) {
          continue;
        }
        int send = rte_eth_tx_burst(port, 0, &mbuf, 1);
        if (send > 0) {
          // success
        } else {
          // The drop can't be memory aligned in cuda object.
          // We need to drop at cpp file.
          // Although, it's not that necessary to drop it.
        }
      }
    } else {
      if (self_batch->host_dst_ports_burst[i] > (uint8_t) num_of_eth_devs) {
        // Drop out, non configured port.
        continue;
      }
      int send = rte_eth_tx_burst(self_batch->host_dst_ports_burst[i], 0, &mbuf, 1);
      if (send > 0) {
        // success
      } else {
        // drop
      }
    }
  }
  self_batch->busy = false;
  return 0;
}

CudaASyncLCoreFunction::CudaASyncLCoreFunction(uint8_t _port_id, unsigned int _num_of_eth_devs,
                                               std::vector<ether_addr> *_mac_addresses_ptr, IPv4RuleEntry *_lpm_table_ptr)
        : port_id(_port_id), num_of_eth_devs(_num_of_eth_devs), mac_addresses_ptr(_mac_addresses_ptr),
          lpm_table_ptr(_lpm_table_ptr) {
  // Do nothing
}

} // namespace cu
} // namespace gpuflow

