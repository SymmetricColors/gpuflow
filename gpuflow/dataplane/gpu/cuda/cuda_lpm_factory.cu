#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include "cuda_lpm_factory.h"
#include <rte_ip.h>
#include <iostream>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>

namespace gpuflow {
namespace cu {

static inline void CudaMallocWithFailOver(void **predicate, size_t size, const char *predicate_type) {
  if (hipMalloc(predicate, size) != hipSuccess) {
    std::cerr << "Device memory allocation on " << predicate_type << " failed, abort." << std::endl;
    exit(1);
  }
}

__global__ void InitLPMTable(IPv4RuleEntry *ipv4_tbl_24) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  ipv4_tbl_24[idx].next_hop = 254;
  ipv4_tbl_24[idx].valid_flag = false;
  ipv4_tbl_24[idx].depth = 0;
  ipv4_tbl_24[idx].external_flag = false;
}

__global__ void InitIPv6LPMTable(IPv6RuleEntry *ipv6_tbl_24) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  ipv6_tbl_24[idx].next_hop = 254;
  ipv6_tbl_24[idx].valid_flag = false;
  ipv6_tbl_24[idx].depth = 0;
  ipv6_tbl_24[idx].external_flag = false;
}

// Create LPM Table
int IPv4LPMFactory::CreateLPMTable() {
  // Allocate lpm table sizes
  CudaMallocWithFailOver((void **)&IPv4TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv4RuleEntry), "IPv4TBL24");
  unsigned long num_of_threads = 2048;
  InitLPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv4TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized lpm entries" << std::endl;
  return 0;
}

// Create IPv6 LPM Table
int IPv6LPMFactory::CreateLPMTable() {
  // Allocate ipv6 lpm table size
  CudaMallocWithFailOver((void **)&IPv6TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv6RuleEntry), "IPv6TBL24");
  unsigned long num_of_threads = 2048;
  InitIPv6LPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv6TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized ipv6 lpm entries" << std::endl;
  return 0;
};

__global__ void SetupRuleEntry(IPv4RuleEntry *ipv4_tbl_24, unsigned long int start, uint8_t next_hop, uint8_t depth) {
  int idx = threadIdx.x;
  ipv4_tbl_24[start + idx].next_hop = next_hop;
  ipv4_tbl_24[start + idx].valid_flag = true;
  ipv4_tbl_24[start + idx].external_flag = false;
  ipv4_tbl_24[start + idx].depth = depth;
  printf("Setup lpm rule entry! index : %lu, next_hop : %d\n", (start + idx), next_hop);
}

int IPv4LPMFactory::AddLPMRule(uint32_t ipv4_address, uint8_t depth, uint8_t next_hop) {
  if (depth > MAX_DEPTH) {
    std::cerr << "Currently, we are not support the tbl 8 secondary search" << std::endl;
    exit(1);
  }

  // FIXME: Back to 24 mask
  unsigned long int start = ipv4_address >> 16 ;
  unsigned long int end = (ipv4_address >> 16) + 1;

  SetupRuleEntry<<<1, end-start>>>(IPv4TBL24, start, next_hop, depth);
  hipDeviceSynchronize();
  return 0;
}

} // namespace cu
} // namespace gpuflow
