#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include "cuda_lpm_factory.h"
#include <rte_ip.h>
#include <iostream>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>

namespace gpuflow {
namespace cu {

static inline void CudaMallocWithFailOver(void **predicate, size_t size, const char *predicate_type) {
  if (hipMalloc(predicate, size) != hipSuccess) {
    std::cerr << "Device memory allocation on " << predicate_type << " failed, abort." << std::endl;
    exit(1);
  }
}

__global__ void InitLPMTable(IPv4RuleEntry *ipv4_tbl_24) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  ipv4_tbl_24[idx].next_hop = 254;
  ipv4_tbl_24[idx].valid_flag = false;
  ipv4_tbl_24[idx].depth = 0;
  ipv4_tbl_24[idx].external_flag = false;
}

__global__ void InitIPv6LPMTable(IPv6RuleEntry *ipv6_tbl_24) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  ipv6_tbl_24[idx].next_hop = 254;
  ipv6_tbl_24[idx].valid_flag = false;
  ipv6_tbl_24[idx].depth = 0;
  ipv6_tbl_24[idx].external_flag = false;
  ipv6_tbl_24[idx].tbl8_ptr = nullptr;
}

// Create LPM Table
int IPv4LPMFactory::CreateLPMTable() {
  // Allocate lpm table sizes
  CudaMallocWithFailOver((void **)&IPv4TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv4RuleEntry), "IPv4TBL24");
  unsigned long num_of_threads = 2048;
  InitLPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv4TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized lpm entries" << std::endl;
  return 0;
}

// Create IPv6 LPM Table
int IPv6LPMFactory::CreateLPMTable() {
  // Allocate ipv6 lpm table size
  CudaMallocWithFailOver((void **)&IPv6TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv6RuleEntry), "IPv6TBL24");
  unsigned long num_of_threads = 2048;
  InitIPv6LPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv6TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized ipv6 lpm entries" << std::endl;
  return 0;
};

__global__ void SetupRuleEntry(IPv4RuleEntry *ipv4_tbl_24, unsigned long int start, uint8_t next_hop, uint8_t depth) {
  int idx = threadIdx.x;
  ipv4_tbl_24[start + idx].next_hop = next_hop;
  ipv4_tbl_24[start + idx].valid_flag = true;
  ipv4_tbl_24[start + idx].external_flag = false;
  ipv4_tbl_24[start + idx].depth = depth;
  printf("Setup lpm rule entry! index : %lu, next_hop : %d\n", (start + idx), next_hop);
}

int IPv4LPMFactory::AddLPMRule(uint32_t ipv4_address, uint8_t depth, uint8_t next_hop) {
  if (depth > MAX_DEPTH) {
    std::cerr << "Currently, we are not support the tbl 8 secondary search" << std::endl;
    exit(1);
  }

  // FIXME: Back to 24 mask
  unsigned long int start = ipv4_address >> 16 ;
  unsigned long int end = (ipv4_address >> 16) + 1;

  SetupRuleEntry<<<1, end-start>>>(IPv4TBL24, start, next_hop, depth);
  hipDeviceSynchronize();
  return 0;
}

__global__ void SetupIPv6RuleEntry(IPv6RuleEntry *ipv6_tbl_24, unsigned long int start, uint8_t next_hop, uint8_t depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ipv6_tbl_24[start + idx].valid_flag && (ipv6_tbl_24[start + idx].depth > depth)) {
    // There's an existed rule and longer, abort this update
  } else {
    // Add the new rule
    // TODO: Currently, we only handle with the case of depth <= 24. The case of depth > 24 using the tbl8 table has not been implemented.
    ipv6_tbl_24[start + idx].next_hop = next_hop;
    ipv6_tbl_24[start + idx].valid_flag = true;
    ipv6_tbl_24[start + idx].external_flag = false;
    ipv6_tbl_24[start + idx].depth = depth;
    ipv6_tbl_24[start + idx].tbl8_ptr = nullptr;
    printf("Setup ipv6 lpm rule entry! index: %lu, next_hop: %d\n", (start + idx), next_hop);
  }
}

int IPv6LPMFactory::AddLPMRule(uint8_t *ipv6_address, uint8_t depth, uint8_t next_hop) {
  if(depth > MAX_DEPTH) {
    std::cerr << "The depth can't be longer than 128" << std::endl;
    exit(1);
  }

  // Calculate the distance
  unsigned long int distance = 1;
  for (unsigned long int i = 0; i < (24 - depth); i++) {
    distance *= 2;
  }

  unsigned long int start;
  if (depth == 24) {
    // The depth is exactly 24

    // Calculate the start
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | ipv6_address[2];
    if ((start + distance) >= (1 << 24)) {
      // overflow
      distance = (1 << 24) - start;
    }

  } else if (depth < 24 && depth >= 16) {
    // The depth is in the range of 16-23.

    // Calculate the  start
    unsigned long int right_shift = ipv6_address[2] >> (24 - depth); 
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | right_shift << (24 - depth);

  } else if (depth < 16 && depth >= 8) {
    // The depth is in the range of 8-15

    // Calculate the start
    unsigned long int right_shift = ipv6_address[1] >> (16 - depth);
    start = ipv6_address[0] << 16 | right_shift << (24 - depth);

  } else if (depth < 8) {
    // The depth is in the range of 0-7

    // Calculate the start
    unsigned long int right_shift = ipv6_address[0] >> (8 -depth);
    start = right_shift << (24 - depth);
      
  } else {
    // TODO: Handling when depth is larger than 24
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | ipv6_address[2];

  }

  unsigned long num_of_threads = 1024;
  if (distance <= num_of_threads) {
    SetupIPv6RuleEntry<<<1, distance>>>(IPv6TBL24, start, next_hop, depth); 
  } else {
    SetupIPv6RuleEntry<<<distance/num_of_threads, num_of_threads>>>(IPv6TBL24, start, next_hop, depth);
  }
  hipDeviceSynchronize();
  return 0;
}

} // namespace cu
} // namespace gpuflow
