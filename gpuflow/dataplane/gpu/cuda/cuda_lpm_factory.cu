#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include "cuda_lpm_factory.h"
#include <rte_ip.h>
#include <iostream>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>

namespace gpuflow {
namespace cu {

static inline void CudaMallocWithFailOver(void **predicate, size_t size, const char *predicate_type) {
  if (hipMalloc(predicate, size) != hipSuccess) {
    std::cerr << "Device memory allocation on " << predicate_type << " failed, abort." << std::endl;
    exit(1);
  }
}

template <typename IPvxRuleEntry>
__global__ void InitIPvxLPMTable(IPvxRuleEntry *ipvx_tbl_ptr) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  ipvx_tbl_ptr[idx].next_hop = 254;
  ipvx_tbl_ptr[idx].valid_flag = false;
  ipvx_tbl_ptr[idx].depth = 0;
  ipvx_tbl_ptr[idx].external_flag = false;
  ipvx_tbl_ptr[idx].tbl8_ptr = nullptr;
}

// Create LPM Table
int IPv4LPMFactory::CreateLPMTable() {
  // Allocate lpm table sizes
  CudaMallocWithFailOver((void **)&IPv4TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv4RuleEntry), "IPv4TBL24");
  unsigned long num_of_threads = 2048;
  InitIPvxLPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv4TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized lpm entries" << std::endl;
  return 0;
}

// Create IPv6 LPM Table
int IPv6LPMFactory::CreateLPMTable() {
  // Allocate ipv6 lpm table size
  CudaMallocWithFailOver((void **)&IPv6TBL24, MAX_LPM_ROUTING_RULES * sizeof(IPv6RuleEntry), "IPv6TBL24");
  unsigned long num_of_threads = 2048;
  InitIPvxLPMTable<<<MAX_LPM_ROUTING_RULES/num_of_threads, num_of_threads>>>(IPv6TBL24);
  hipDeviceSynchronize();
  std::cout << "Initialized ipv6 lpm entries" << std::endl;
  return 0;
};

__global__ void SetupRuleEntry(IPv4RuleEntry *ipv4_tbl_24, unsigned long int start, uint8_t next_hop, uint8_t depth) {
  int idx = threadIdx.x;
  ipv4_tbl_24[start + idx].next_hop = next_hop;
  ipv4_tbl_24[start + idx].valid_flag = true;
  ipv4_tbl_24[start + idx].external_flag = false;
  ipv4_tbl_24[start + idx].depth = depth;
  printf("Setup lpm rule entry! index : %lu, next_hop : %d\n", (start + idx), next_hop);
}

int IPv4LPMFactory::AddLPMRule(uint32_t ipv4_address, uint8_t depth, uint8_t next_hop) {
  if (depth > MAX_DEPTH) {
    std::cerr << "Currently, we are not support the tbl 8 secondary search" << std::endl;
    exit(1);
  }

  // FIXME: Back to 24 mask
  unsigned long int start = ipv4_address >> 16 ;
  unsigned long int end = (ipv4_address >> 16) + 1;

  SetupRuleEntry<<<1, end-start>>>(IPv4TBL24, start, next_hop, depth);
  hipDeviceSynchronize();
  return 0;
}

__global__ void SetupIPv6RuleEntry(IPv6RuleEntry *ipv6_tbl_24, unsigned long int start, uint8_t next_hop, uint8_t depth,
                                   IPv6RuleEntry *ipv6_tbl_8) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ipv6_tbl_24[start + idx].valid_flag && (ipv6_tbl_24[start + idx].depth > depth)) {
    // There's an existed rule and longer, abort this update
  } else {
    // Add new rule with a pointer points to the tbl8 table.
    ipv6_tbl_24[start + idx].next_hop = next_hop;
    ipv6_tbl_24[start + idx].valid_flag = true;
    ipv6_tbl_24[start + idx].depth = depth;
    ipv6_tbl_24[start + idx].tbl8_ptr = ipv6_tbl_8;
    if (ipv6_tbl_8 != nullptr) ipv6_tbl_24[start + idx].external_flag = true;
    printf("Setup ipv6 lpm rule entry! index: %lu, next_hop: %d\n", (start + idx), next_hop);
  }
}

__global__ void SetupIPv6TBL8RuleEntry(IPv6RuleEntry *ipv6_tbl_8, IPv6RuleEntry *next_tbl_8, unsigned long start,
                                       uint8_t next_hop, uint8_t depth) {
  int idx = threadIdx.x;
  ipv6_tbl_8[start + idx].next_hop = next_hop;
  ipv6_tbl_8[start + idx].valid_flag = true;
  ipv6_tbl_8[start + idx].depth = depth;
  ipv6_tbl_8[start + idx].tbl8_ptr = next_tbl_8;
  if (next_tbl_8 != nullptr) ipv6_tbl_8[start + idx].external_flag = true;
}

int IPv6LPMFactory::AddLPMRule(uint8_t *ipv6_address, uint8_t depth, uint8_t next_hop) {
  if(depth > MAX_DEPTH) {
    std::cerr << "The depth can't be longer than 128" << std::endl;
    exit(1);
  }

  // Calculate the distance
  unsigned long int distance = 1;
  if (depth <= 24) {
    for (unsigned long int i = 0; i < (24 - depth); i++) {
      distance *= 2;
    }
  }

  IPv6RuleEntry *ipv6_tbl8_ptrs[13] = { nullptr };
  unsigned long int start = 0;

  if (depth == 24) {
    // The depth is exactly 24

    // Calculate the start
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | ipv6_address[2];
    if ((start + distance) >= (1 << 24)) {
      // overflow
      distance = (1 << 24) - start;
    }

  } else if (depth < 24 && depth >= 16) {
    // The depth is in the range of 16-23.

    // Calculate the  start
    unsigned long int right_shift = ipv6_address[2] >> (24 - depth);
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | right_shift << (24 - depth);

  } else if (depth < 16 && depth >= 8) {
    // The depth is in the range of 8-15

    // Calculate the start
    unsigned long int right_shift = ipv6_address[1] >> (16 - depth);
    start = ipv6_address[0] << 16 | right_shift << (24 - depth);

  } else if (depth < 8) {
    // The depth is in the range of 0-7

    // Calculate the start
    unsigned long int right_shift = ipv6_address[0] >> (8 -depth);
    start = right_shift << (24 - depth);
      
  } else {
    // Handling when depth is larger than 24
    start = ipv6_address[0] << 16 | ipv6_address[1] << 8 | ipv6_address[2];
   
    // Calculate the number of tbl8 table 
    int tbl_8_number = depth % 8 ? ((depth - 24) / 8) + 1 : (depth - 24) / 8;
    
    // Calculate the distance of the last tbl8 table. 
    int tbl_8_last_table_distance = 1;
    for (int i = 0; i < (depth % 8); i++) {
      tbl_8_last_table_distance *= 2;
    }
   
    // Malloc memory in the cuda device,  initialize them and set up the rules. 
    for (int i = 0; i < tbl_8_number; i++) {
      CudaMallocWithFailOver((void **)&ipv6_tbl8_ptrs[i], 256 * sizeof(IPv6RuleEntry), "IPv6TBL8");
      InitIPvxLPMTable<<<1, 256>>>(ipv6_tbl8_ptrs[i]);
      if (i < (tbl_8_number - 1)) {
        // Not the last tbl8 table
        SetupIPv6TBL8RuleEntry<<<1, 1>>>(ipv6_tbl8_ptrs[i], ipv6_tbl8_ptrs[i + 1], ipv6_address[3 + i], next_hop, depth);
      } else {
        // The last tbl8 table
        SetupIPv6TBL8RuleEntry<<<1, tbl_8_last_table_distance>>>(ipv6_tbl8_ptrs[i], nullptr, ipv6_address[3 + i], next_hop, depth);
      }
    }
  }

  unsigned long num_of_threads = 1024;
  if (distance <= num_of_threads) {
    // TBL24 does not have to point to the tbl8 table.
    SetupIPv6RuleEntry<<<1, distance>>>(IPv6TBL24, start, next_hop, depth, ipv6_tbl8_ptrs[0]);
  } else {
    // FIXME: Not correct sizes
    // Assume that distance == 1025 ?
    SetupIPv6RuleEntry<<<distance/num_of_threads, num_of_threads>>>(IPv6TBL24, start, next_hop, depth, nullptr);
  }
  hipDeviceSynchronize();
  return 0;
}

} // namespace cu
} // namespace gpuflow
